
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

//add function for two arrays
//*x means point to memory address of x
//Global represents that available to do on GPU
__global__
void add(int n, float *x, float *y)
{
    for (int i =0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void){
    int N = 1<<20; //1M (1,048,576) elements, shift 1 over 20 times in binary
    float *x, *y;
    //to print (std::cout << ""<<) 
    std::cout << "N value:" << N << std::endl;


    //Allocate unified Memory -accesible from CPU and GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    //initialize arrays
    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //run kernal on elemants on CPU
    //add(N,x,y);

    //run kernal on GPU
    add<<<1, 1>>>(N, x, y);

    //wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    //check for errors, all should be 3 (1.0 + 2.0)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    std::cout << "Max Error: " << maxError << std::endl;

    //freee up GPU memeory
    hipFree(x);
    hipFree(y);

    //Free up  cpu memory
    // delete [] x;
    // delete [] y;

    return 0;

}